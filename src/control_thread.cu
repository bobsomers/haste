#include "hip/hip_runtime.h"
#include "control_thread.cu.h"

ControlThread::ControlThread(int device_id, uint16_t start, uint16_t width) :
    Thread(),
    _device_id(device_id),
    _start(start),
    _width(width),
    _num_threads(0),
    _num_blocks(0),
    _layer_buffers(NULL),
    _rand_states(NULL),
    _meta_chunk(NULL),
    _light_list(NULL),
    _mat_list(NULL),
    _obj_chunk(NULL),
    _total_rays(0),
    _ray_chunk(NULL),
    _final(NULL) {
    // empty
}

ControlThread::~ControlThread() {
    // release image buffer if it was allocated
    if (_final != NULL) {
        free(_final);
        _final = NULL;
    }
}

void ControlThread::Run() {
    // initialize the device for this thread
    printf("[%d] Initializing device...\n", _device_id);
    InitializeDevice();
    
    // allocate buffers on the gpu
    printf("[%d] Allocating buffers...\n", _device_id);
    AllocateBuffers();

    // copy data to the gpu
    printf("[%d] Copying data to device...\n", _device_id);
    CopyDataToDevice();

    // generate primary rays
    printf("[%d] Generating primary rays...\n", _device_id);
    GeneratePrimaryRays();

    // stream the rays down to the device
    printf("[%d] Starting trace...\n", _device_id);
    _total_rays = 0;
    uint32_t packet_size = _num_threads * _num_blocks;
    do {
        // extract next packet of rays
        uint32_t num_rays = (_ray_queue.size() >= packet_size) ? packet_size : _ray_queue.size();
        Ray *chunk = (Ray *)malloc(sizeof(Ray) * num_rays);
        for (uint32_t i = 0; i < num_rays; i++) {
            memcpy(&(chunk[i]), _ray_queue.front(), sizeof(Ray));
            free(_ray_queue.front());
            _ray_queue.pop();
        }

        // prepare the rays for tracing
        CUDA_SAFE_CALL(hipMemcpy(_ray_chunk, chunk, sizeof(Ray) * num_rays, hipMemcpyHostToDevice));
        RayPacket packet = {_ray_chunk, num_rays};

        // launch the kernel
        device::RayTrace<<<_num_blocks, _num_threads>>>(packet);

        // copy output rays back from the device
        // TODO

        // inject output rays into the ray queue
        // TODO

        // clean up
        free(chunk);
        
        // increment ray cast count
        _total_rays += num_rays;
    } while (_ray_queue.size() > 0);

    // collapse layer buffers
    // TODO
    
    // copy the base layer buffer into host memory as the final image
    _final = GetLayerBuffer(0);
    
    // remove the scene and layer buffers from the gpu
    printf("[%d] Cleaning up...\n", _device_id);
    CleanUp();
    
    // control thread for this device finished
    printf("[%d] Finished.\n", _device_id);
}

void ControlThread::InitializeDevice() {
    // set the device
    CUDA_SAFE_CALL(hipSetDevice(_device_id));
    
    // query the device to get the number of threads and blocks we should
    // launch kernels with
    hipDeviceProp_t device_prop;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&device_prop, _device_id));
    _num_threads = device_prop.maxThreadsPerBlock / 2;
    _num_blocks = device_prop.multiProcessorCount * 2;
    
    // set the stack size to 16 KB for random number generation with CURAND
    cudaThreadSetLimit(hipLimitStackSize, 16384);
    
    // allocate space for the device for the random number generator states
    uint32_t packet_size = _num_threads * _num_blocks;
    CUDA_SAFE_CALL(hipMalloc<hiprandState>(&_rand_states, sizeof(hiprandState) * packet_size));
    
    // initialize the random number generator states
    device::InitRandomness<<<_num_blocks, _num_threads>>>(time(NULL), _rand_states);
}

void ControlThread::AllocateBuffers() {
    // allocate one huge chunk for all the layer buffers
    CUDA_SAFE_CALL(hipMalloc<float3>(&_layer_buffers, sizeof(float3) * _width * host::render.size.y * host::render.max_bounces));
    
    // zero out each layer buffer by copying zeros into the device memory for each layer
    float3 *zeroed = (float3 *)malloc(sizeof(float3) * _width * host::render.size.y);
    memset(zeroed, 0, sizeof(float3) * _width * host::render.size.y);
    for (uint64_t i = 0; i < host::render.max_bounces; i++) {
        uint64_t layer_offset = sizeof(float3) * _width * host::render.size.y * i;
        float3 *layer = (float3 *) ((uint64_t)(_layer_buffers) + layer_offset);
        CUDA_SAFE_CALL(hipMemcpy(layer, zeroed, sizeof(float3) * _width * host::render.size.y, hipMemcpyHostToDevice));
    }
    free(zeroed);
    
    // allocate space for the meta chunk on the device
    CUDA_SAFE_CALL(hipMalloc<MetaObject>(&_meta_chunk, sizeof(MetaObject) * host::num_objs));
    
    // allocate space for the light list on the device
    CUDA_SAFE_CALL(hipMalloc<LightObject>(&_light_list, sizeof(LightObject) * host::num_lights));
    
    // allocate space for the material list on the device
    CUDA_SAFE_CALL(hipMalloc<Material>(&_mat_list, sizeof(Material) * host::num_mats));
    
    // allocate space for the object chunk on the device
    CUDA_SAFE_CALL(hipMalloc(&_obj_chunk, host::obj_chunk_size));
    
    // allocate space for the ray chunk on the device
    uint32_t packet_size = _num_threads * _num_blocks;
    CUDA_SAFE_CALL(hipMalloc<Ray>(&_ray_chunk, sizeof(Ray) * packet_size));
}

void ControlThread::CopyDataToDevice() {
    // copy the meta chunk to the device
    CUDA_SAFE_CALL(hipMemcpy(_meta_chunk, host::meta_chunk, sizeof(MetaObject) * host::num_objs, hipMemcpyHostToDevice));
    
    // copy the light list to the device
    CUDA_SAFE_CALL(hipMemcpy(_light_list, host::light_list, sizeof(LightObject) * host::num_lights, hipMemcpyHostToDevice));
   
    // copy the material list to the device
    CUDA_SAFE_CALL(hipMemcpy(_mat_list, host::mat_list, sizeof(Material) * host::num_mats, hipMemcpyHostToDevice));

    // copy the object chunk to the device
    CUDA_SAFE_CALL(hipMemcpy(_obj_chunk, host::obj_chunk, host::obj_chunk_size, hipMemcpyHostToDevice));
    
    // copy trace parameters to device constant memory
    TraceParams params;
    params.render = host::render;
    params.start = _start;
    params.width = _width;
    params.meta_chunk = _meta_chunk;
    params.num_objs = host::num_objs;
    params.light_list = _light_list;
    params.num_lights = host::num_lights;
    params.mat_list = _mat_list;
    params.num_mats = host::num_mats;
    params.obj_chunk = _obj_chunk;
    params.layer_buffers = _layer_buffers;
    params.rand_states = _rand_states;
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("device::PARAMS"), &params, sizeof(TraceParams)));
}

void ControlThread::CleanUp() {
    // free device layer buffer memory
    if (_layer_buffers != NULL) {
	    CUDA_SAFE_CALL(hipFree(_layer_buffers));
	    _layer_buffers = NULL;
    }
    
    // free device scene memory
    if (_meta_chunk != NULL) {
    	CUDA_SAFE_CALL(hipFree(_meta_chunk));
    	_meta_chunk = NULL;
   	}
   	if (_light_list != NULL) {
   		CUDA_SAFE_CALL(hipFree(_light_list));
   		_light_list = NULL;
   	}
    if (_mat_list != NULL) {
        CUDA_SAFE_CALL(hipFree(_mat_list));
        _mat_list = NULL;
    }
    if (_obj_chunk != NULL) {
    	CUDA_SAFE_CALL(hipFree(_obj_chunk));
    	_obj_chunk = NULL;
   	}
   	
   	// free device ray chunk memory
    if (_ray_chunk != NULL) {
        CUDA_SAFE_CALL(hipFree(_ray_chunk));
        _ray_chunk = NULL;
    }
   	
   	// free device randomness state
   	if (_rand_states != NULL) {
        CUDA_SAFE_CALL(hipFree(_rand_states));
        _rand_states = NULL;
    }
}

void ControlThread::GeneratePrimaryRays() {
    // seed the random number generator
    srand(time(NULL));

    // compute bottom left of screen space extents
    float l = host::camera.aspect / -2.0f;
    float b = -0.5f;

    // compute camera's gaze vector
    float3 w = normalize(host::camera.look - host::camera.eye);

    // now compute the camera's up vector (not factoring in rotation yet)
    float3 temp = normalize(cross(w, host::camera.up));
    float3 v = normalize(cross(temp, w));

    // compute the point of the tip of the up vector
    float3 v_pt = host::camera.eye + v;
    
    // now rotate this point around the gaze vector
    // huge thanks to http://www.blitzbasic.com/Community/posts.php?topic=57616#645017 for this
    float3 rotated_v_pt;
    rotated_v_pt.x = w.x * (w.x * v_pt.x + w.y * v_pt.y + w.z * v_pt.z) + (v_pt.x * (w.y * w.y + w.z * w.z) - w.x * (w.y * v_pt.y + w.z * v_pt.z)) * cosf(host::camera.rotation * (float)M_PI / 180.0f) + (-w.z * v_pt.y + w.y * v_pt.z) * sinf(host::camera.rotation * (float)M_PI / 180.0f);
    rotated_v_pt.y = w.y * (w.x * v_pt.x + w.y * v_pt.y + w.z * v_pt.z) + (v_pt.y * (w.x * w.x + w.z * w.z) - w.y * (w.x * v_pt.x + w.z * v_pt.z)) * cosf(host::camera.rotation * (float)M_PI / 180.0f) + (w.z * v_pt.x - w.x * v_pt.z) * sinf(host::camera.rotation * (float)M_PI / 180.0f);
    rotated_v_pt.z = w.z * (w.x * v_pt.x + w.y * v_pt.y + w.z * v_pt.z) + (v_pt.z * (w.x * w.x + w.y * w.y) - w.z * (w.x * v_pt.x + w.y * v_pt.y)) * cosf(host::camera.rotation * (float)M_PI / 180.0f) + (-w.y * v_pt.x + w.x * v_pt.y) * sinf(host::camera.rotation * (float)M_PI / 180.0f);

    // recalculate the rotated up vector by subtracting off the eye position
    v = normalize(rotated_v_pt - host::camera.eye);

    // compute camera's u vector
    float3 u = normalize(cross(w, v));

    // loop over all pixels in this slice (and each antialiasing cell per pixel)
    for (uint16_t x = _start; x < _start + _width; x++) {
        for (uint16_t y = 0; y < host::render.size.y; y++) {
            for (uint32_t i = 0; i < host::render.antialiasing; i++) {
                for (uint32_t j = 0; j < host::render.antialiasing; j++) {
                    float us = 0.0f;
                    float vs = 0.0f;
                    float ws = 0.0f;
                    float contrib = 0.0f;
                    
                    // calculate screen space uvw
                    if (host::render.antialiasing <= 1) { // no antialiasing
                        us = l + (host::camera.aspect * (x + 0.5f) / host::render.size.x);
                        vs = b + (1.0f * (y + 0.5f) / host::render.size.y);
                        ws = 1.0f;      
                        contrib = 1.0f;              
                    } else {
                        float cell_size = 1.0f / host::render.antialiasing;
                        float rand_offset = (float)rand() / RAND_MAX;
                        us = l + (host::camera.aspect * (x + (i * cell_size) + (rand_offset * cell_size)) / host::render.size.x);
                        rand_offset = (float)rand() / RAND_MAX;
                        vs = b + (1.0f * (y + (j * cell_size) + (rand_offset * cell_size)) / host::render.size.y);
                        ws = 1.0f;
                        contrib = 1.0f / (host::render.antialiasing * host::render.antialiasing);
                    }

                    // convert screen space point to world coords
                    float3 screen_pt = host::camera.eye +
                                       (u * make_float3(us, us, us)) +
                                       (v * make_float3(vs, vs, vs)) +
                                       (w * make_float3(ws, ws, ws));

                    // create ray
                    Ray *ray = (Ray *)malloc(sizeof(Ray));
                    ray->origin = host::camera.eye;
                    ray->direction = normalize(screen_pt - host::camera.eye);
                    ray->contrib = contrib;
                    ray->layer = 0;
                    ray->pixel = make_ushort2(x, y);
                    ray->unibounce = false;

                    // push it into the ray queue
                    _ray_queue.push(ray);
                }
            }
        }
    }
}

float3 *ControlThread::GetLayerBuffer(uint64_t layer) {
    // compute source address
	uint64_t offset = sizeof(float3) * _width * host::render.size.y * layer;
	void *src = (void *)((uint64_t)_layer_buffers + offset);

	// allocate space on the host for the destination buffer
	float3 *dest = (float3 *)malloc(sizeof(float3) * _width * host::render.size.y);
    CUDA_SAFE_CALL(hipMemcpy(dest, src, sizeof(float3) * _width * host::render.size.y, hipMemcpyDeviceToHost));
    
    return dest;
}
