#include "hip/hip_runtime.h"
#include "raytrace.cu.h"

namespace device {
    __device__ const float EPSILON = 0.0001f;
    __constant__ TraceParams PARAMS;
}

// ===== math functions =====

__device__ float3 device::operator+(const float3 &lhs, const float3 &rhs) {
    return make_float3(lhs.x + rhs.x,
                        lhs.y + rhs.y,
                        lhs.z + rhs.z);
}

__device__ float3 device::operator-(const float3 &lhs, const float3 &rhs) {
    return make_float3(lhs.x - rhs.x,
                        lhs.y - rhs.y,
                        lhs.z - rhs.z);
}

__device__ float3 device::operator*(const float3 &lhs, const float3 &rhs) {
    return make_float3(lhs.x * rhs.x,
                        lhs.y * rhs.y,
                        lhs.z * rhs.z);
}

__device__ float3 device::operator/(const float3 &lhs, const float3 &rhs) {
    return make_float3(lhs.x / rhs.x,
                        lhs.y / rhs.y,
                        lhs.z / rhs.z);
}

__device__ float3 device::operator/(const float3 &lhs, const float &wgt) {
    return make_float3(lhs.x / wgt,
                        lhs.y / wgt,
                        lhs.z / wgt);
}

__device__ float3 device::operator*(const float3 &lhs, const float &wgt) {
    return make_float3(lhs.x * wgt,
                        lhs.y * wgt,
                        lhs.z * wgt);
}

__device__ float device::dot(const float3 &lhs, const float3 &rhs) {
    return (lhs.x * rhs.x) + (lhs.y * rhs.y) + (lhs.z * rhs.z);
}

__device__ float3 device::cross(const float3 &lhs, const float3 &rhs) {
    return make_float3((lhs.y * rhs.z) - (lhs.z * rhs.y),
                        (lhs.z * rhs.x) - (lhs.x * rhs.z),
                        (lhs.x * rhs.y) - (lhs.y * rhs.x));
}

__device__ float device::length(const float3 &v) {
    return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ float device::distance(const float3 &a, const float3 &b) {
    return length(a - b);
}

__device__ float3 device::normalize(const float3 &v) {
    float len = length(v);
    return make_float3(v.x / len, v.y / len, v.z / len);
}

__device__ float3 device::evaluate(Ray *ray, float t) {
    return ray->origin + (ray->direction * make_float3(t, t, t));
}

__device__ float device::triarea(const float3 &a, const float3 &b, const float3 &c) {
    // heron's formula
    float i = distance(a, b);
    float j = distance(a, c);
    float k = distance(b, c);
    float s = (i + j + k) / 2.0f;    
    return sqrtf(s * (s - i) * (s - j) * (s - k));
}

// ===== randomness helpers =====

__device__ hiprandState device::GetRandState() {
    return PARAMS.rand_states[threadIdx.x + blockIdx.x * blockDim.x];
}

// ===== normal functions =====

__device__ float3 device::Normal(Sphere *sphere, const float3 &point) {
    float3 normal = point - sphere->position;
    return normalize(normal);
}

__device__ float3 device::Normal(Plane *plane, const float3 &point) {
    return normalize(plane->normal);
}

__device__ float3 device::Normal(Triangle *triangle, const float3 &point) {
    /*
    // returns the smooth normal of the triangle (interpolated from vertex normals)
    // this uses the ratios of the areas of the three sub-triangles created by
    // the point to weight the normal contribution
    // thanks to http://74.86.81.120/Community/posts.php?topic=57617
    float area_a = triarea(point, triangle->vertex2, triangle->vertex3);
    float area_b = triarea(triangle->vertex1, point, triangle->vertex3);
    float area_c = triarea(triangle->vertex1, triangle->vertex2, point);
    float area_total = area_a + area_b + area_c;
    
    // calculate weighting coefficients
    float weight1 = area_a / area_total;
    float weight2 = area_b / area_total;
    float weight3 = area_c / area_total;
    
    // interpolate the normal
    return normalize(make_float3((triangle->normal1.x * weight1) + (triangle->normal2.x * weight2) + (triangle->normal3.x * weight3),
                                  (triangle->normal1.y * weight1) + (triangle->normal2.y * weight2) + (triangle->normal3.y * weight3),
                                  (triangle->normal1.z + weight1) + (triangle->normal2.z * weight2) + (triangle->normal3.z * weight3)));
    */

	#define X1 (float)triangle->vertex1.x
	#define X2 (float)triangle->vertex2.x
	#define X3 (float)triangle->vertex3.x

	#define Y1 (float)triangle->vertex1.y
	#define Y2 (float)triangle->vertex2.y
	#define Y3 (float)triangle->vertex3.y

	#define Z1 (float)triangle->vertex1.z
	#define Z2 (float)triangle->vertex2.z
	#define Z3 (float)triangle->vertex3.z

/*
	float u = ((float) ( (Y2 - Y3) * (point.x - X3) ) + ( (X3 - X2) * (point.y - Y3) ) ) / ( ( (Y2 - Y3) * (X1 - X3) ) + ( (X3 - X2) * (Y1 - Y3) ) );

	float v = ((float) ( (Y3 - Y1) * (point.x - X3) ) + ( (X1 - X3) * (point.y - Y3) ) ) / ( ( (Y3 - Y1) * (X2 - X3) ) + ( (X1 - X3) * (Y2 - Y3) ) );

	float w = 1.f - (u + v);
*/
/*
	float3 U = make_float3(X1 - X2, Y1 - Y2, Z1 - Z2);
	float3 V = make_float3(X3 - X2, Y3 - Y2, Z3 - Z2);
	
	float3 N = make_float3(point.x - X2, -Y2, point.z - Z2);

	float dU = sqrt(pow(U.x, 2) + pow(U.y, 2) + pow(U.z, 2));
	float dV = sqrt(pow(V.x, 2) + pow(V.y, 2) + pow(V.z, 2));
	float dN = sqrt(pow(N.x, 2) + pow(N.y, 2) + pow(N.z, 2));

	U = normalize(U);
	N = normalize(N);

	float cost = dot(N, U);
	if(cost < 0.f) cost = 0.f;
	if(cost > 1.f) cost = 1.f;

	float t = acos(cost);

	float distY = 0;
	float distX = 0;

	distX = dN * cos(t);
	distY = dN * sin(t);

	float u = distX / dU;
	float v = distY / dV;
	float w = 1.f - (u + v);
	*/

	float3 abcCross = cross(triangle->vertex2 - triangle->vertex1, triangle->vertex3 - triangle->vertex1);
	float3 pbcCross = cross(triangle->vertex2 - point, triangle->vertex3 - point);
	float3 pcaCross = cross(triangle->vertex3 - point, triangle->vertex1 - point);
	float3 pabCross = cross(triangle->vertex1 - point, triangle->vertex2 - point);

	float3 N = normalize( abcCross );

	float AreaABC = dot(N, abcCross );

	float AreaPBC = dot(N, pbcCross );
	float a = AreaPBC / AreaABC;

	float AreaPCA = dot(N, pcaCross );
	float b = AreaPCA / AreaABC;

	float AreaPAB = dot(N, pabCross );
	float c = AreaPAB / AreaABC;

	//float c = 1.0f - a - b;

	
	

	#undef X1
	#undef X2
	#undef X3

	#undef Y1
	#undef Y2
	#undef Y3

	#undef Z1
	#undef Z2
	#undef Z3

	float weight1 = a;
	float weight2 = b;
	float weight3 = c;

	//triangle->normal1 = normalize(triangle->normal1);
	//triangle->normal2 = normalize(triangle->normal2);
	//triangle->normal3 = normalize(triangle->normal3);

    return normalize(make_float3((triangle->normal1.x * weight1) + (triangle->normal2.x * weight2) + (triangle->normal3.x * weight3),
                                  (triangle->normal1.y * weight1) + (triangle->normal2.y * weight2) + (triangle->normal3.y * weight3),
                                  (triangle->normal1.z * weight1) + (triangle->normal2.z * weight2) + (triangle->normal3.z * weight3)));

	//float3 norm = ((triangle->normal1) * a) * ((triangle->normal2) * b) * ((triangle->normal3) * c);
	//return normalize(norm);
}

__device__ float3 device::Normal(Triangle *triangle) {
    // returns the face normal of the triangle (NOT INTERPOLATED FROM VERTEX NORMALS!)
    float3 AB = triangle->vertex2 - triangle->vertex1;
    float3 AC = triangle->vertex3 - triangle->vertex1;
    return normalize(cross(AB, AC));
}

__device__ float3 device::Normal(Intersection *obj, const float3 &point) {
    // TODO: handle transforms here
    
    switch (obj->type) {
        case SPHERE:
            return Normal((Sphere *)(obj->ptr), point);
            
        case PLANE:
            return Normal((Plane *)(obj->ptr), point);
            
        case TRIANGLE:
            return Normal((Triangle *)(obj->ptr), point);
    }
    
    return make_float3(0.0f, 0.0f, 0.0f);
}

// ===== intersection functions =====

__device__ float device::Intersect(Ray *ray, Sphere *sphere) {
    // calculate quadratic components
    float a = dot(ray->direction, ray->direction);
    float b = dot(ray->direction, (ray->origin - sphere->position)) * 2.0f;
    float c = dot((ray->origin - sphere->position), (ray->origin - sphere->position)) - (sphere->radius * sphere->radius);

    float discr = (b * b) - (4.0f * a * c);
    if (discr > 0.0f) {
        // two intersections, return the closest positive t
        float t1 = ((-1.0f * b) + sqrtf(discr)) / (2.0f * a);
        float t2 = ((-1.0f * b) - sqrtf(discr)) / (2.0f * a);

        if (t1 > EPSILON && t2 > EPSILON) {
            return fminf(t1, t2);
        } else if (t1 < -EPSILON && t2 < -EPSILON) {
            return fmaxf(t1, t2);
        } else {
            return (fabs(t1) < EPSILON) ? t2 : t1;
        }
    } else if (discr == 0.0f) {
        // barely grazes the edge, only a single intersection
        return (-1.0f * b) / (2.0f * a);
    }

    // does not intersect the sphere
    return -1.0f;
}

__device__ float device::Intersect(Ray *ray, Plane *plane) {
    // find p1 (a point on the plane) by using the formula for the point on the plane
    // closest to the origin
    float a = plane->normal.x;
    float b = plane->normal.y;
    float c = plane->normal.z;
    float d = plane->distance;
    float abc_sq = a * a + b * b + c * c;
    float3 p1 = make_float3((a * d) / abc_sq,
                             (b * d) / abc_sq,
                             (c * d) / abc_sq);
                             
    // solve for t
    float denom = dot(ray->direction, plane->normal);
    if (denom != 0.0f) {
        return dot((p1 - ray->origin), plane->normal) / denom;
    }

    // plane and ray are parallel, no intersection
    return -1.0f;
}

__device__ float device::Intersect(Ray *ray, Triangle *triangle) {
    // quick plane intersection test to avoid expensive barycentric test
    float3 N = Normal(triangle);
    float denom = dot(ray->direction, N);
    if (denom == 0.0f) {
        // ray and triangle are in parallel planes, no intersection
        return -1.0f;
    }
    float t = dot((triangle->vertex1 - ray->origin), N) / denom;
    if (t < 0.0f) {
        // we don't care, intersects behind
        return -1.0f;
    }
    float3 p = evaluate(ray, t);

    // compute the matrix members
    float a = triangle->vertex1.x - triangle->vertex2.x;
    float b = triangle->vertex1.y - triangle->vertex2.y;
    float c = triangle->vertex1.z - triangle->vertex2.z;
    float d = triangle->vertex1.x - triangle->vertex3.x;
    float e = triangle->vertex1.y - triangle->vertex3.y;
    float f = triangle->vertex1.z - triangle->vertex3.z;
    float g = ray->direction.x;
    float h = ray->direction.y;
    float i = ray->direction.z;
    float j = triangle->vertex1.x - p.x;
    float k = triangle->vertex1.y - p.y;
    float l = triangle->vertex1.z - p.z;


    // compute the determinant of M
    float detM = a * (e * i - h * f) + b * (g * f - d * i) + c * (d * h - e * g);
    if (detM == 0.0f) {
        // no point in going any further
        return -1.0f;
    }


    // next compute gamma, and check to make sure it's in range (>= 0)
    float gamma = i * (a * k - j * b) + h * (j * c - a * l) + g * (b * l - k * c);
    gamma /= detM;
    if (gamma < 0.0f) {
        return -1.0f;
    }
    
    // lastly, compute beta, and check to make sure it's in range (>= 0)
    float beta = j * (e * i - h * f) + k * (g * f - d * i) + l * (d * h - e * g);
    beta /= detM;
    if (beta < 0.0f) {
        return -1.0f;
    }


    // finally check our beta/gamma combined range to make sure it intersects (beta + gamma <= 1)
    if (beta + gamma > 1.0f) {
        return -1.0f;
    }
    
    // intersection is good!
    return t;
}

__device__ bool device::Intersect(Ray *ray, Intersection *obj) {
    // TODO: transform the ray by the inverse transformation matrix

    switch (obj->type) {
        case SPHERE:
            obj->t = Intersect(ray, (Sphere *)(obj->ptr));
            if (obj->t > EPSILON) return true;
            break;
            
        case PLANE:
            obj->t = Intersect(ray, (Plane *)(obj->ptr));
            if (obj->t > EPSILON) return true;
            break;
            
        case TRIANGLE:
            obj->t = Intersect(ray, (Triangle *)(obj->ptr));
            if (obj->t > EPSILON) return true;
            break;
    }

    return false;
}

__device__ Intersection device::NearestObj(Ray *ray) {
    Intersection closest = {SPHERE, NULL, -1.0f};
 
    // check all objects for intersections
    for (uint64_t i = 0; i < PARAMS.num_objs; i++) {
        Intersection obj;
        obj.type = PARAMS.meta_chunk[i].type;
        if (obj.type == LIGHT) continue; // don't waste time on point lights
        obj.ptr = (void *) ((uint64_t) PARAMS.obj_chunk + PARAMS.meta_chunk[i].offset);
        if (Intersect(ray, &obj)) {
            if (closest.t < 0.0f) {
                closest = obj;
            } else {
                if (obj.t < closest.t) {
                    closest = obj;
                }
            }
        }
    }

    return closest;
}

// ===== accessor functions =====

__device__ float3 device::GetLayerBuffer(ushort2 pixel, uint64_t layer) {
    // shift pixel coord into this slice's buffer space
    ushort2 pxl = make_ushort2(pixel.x - PARAMS.start, pixel.y);
    
    // calculate memory offsets
    uint64_t layer_offset = sizeof(float3) * PARAMS.width * PARAMS.render.size.y * layer;
    uint64_t pixel_offset = sizeof(float3) * (pxl.x + pxl.y * PARAMS.width);
    float3 *clr = (float3 *)((uint64_t)(PARAMS.layer_buffers) + layer_offset + pixel_offset);
    
    return *clr;
}

__device__ void device::SetLayerBuffer(ushort2 pixel, uint64_t layer, float3 color) {
    // shift pixel coord into this slice's buffer space
    ushort2 pxl = make_ushort2(pixel.x - PARAMS.start, pixel.y);

    // calculate memory offsets
    uint64_t layer_offset = sizeof(float3) * PARAMS.width * PARAMS.render.size.y * layer;
    uint64_t pixel_offset = sizeof(float3) * (pxl.x + pxl.y * PARAMS.width);
    float3 *clr = (float3 *)((uint64_t)(PARAMS.layer_buffers) + layer_offset + pixel_offset);
    
    *clr = color; 
}

__device__ void device::BlendWithLayerBuffer(ushort2 pixel, uint64_t layer, float3 color) {
    // shift pixel coord into this slice's buffer space
    ushort2 pxl = make_ushort2(pixel.x - PARAMS.start, pixel.y);
    
    // calculate memory offsets
    uint64_t layer_offset = sizeof(float3) * PARAMS.width * PARAMS.render.size.y * layer;
    uint64_t pixel_offset = sizeof(float3) * (pxl.x + pxl.y * PARAMS.width);
    float *addr;
    
    // red component    
    addr = (float *)((uint64_t)(PARAMS.layer_buffers) + layer_offset + pixel_offset + (0 * sizeof(float)));
    atomicAdd(addr, color.x);
    
    // green component    
    addr = (float *)((uint64_t)(PARAMS.layer_buffers) + layer_offset + pixel_offset + (1 * sizeof(float)));
    atomicAdd(addr, color.y);
    
    // blue component    
    addr = (float *)((uint64_t)(PARAMS.layer_buffers) + layer_offset + pixel_offset + (2 * sizeof(float)));
    atomicAdd(addr, color.z);
}

// ===== shading functions =====

__device__ Material* device::GetMaterial(Intersection *obj) {
    uint64_t mat_id = 0;

    switch (obj->type) {
        case SPHERE:
            mat_id = ((Sphere *)(obj->ptr))->material;
            return &(PARAMS.mat_list[mat_id]);
            
        case PLANE:
            mat_id = ((Plane *)(obj->ptr))->material;
            return &(PARAMS.mat_list[mat_id]);
            
        case TRIANGLE:
            mat_id = ((Triangle *)(obj->ptr))->material;
            return &(PARAMS.mat_list[mat_id]);
    }
    
    return NULL;
}

__device__ float3 device::GetLightColor(LightObject *light) {
    Light *ptlt = NULL;
    Sphere *sphere = NULL;
    Material *mat = NULL;
    
    switch (light->type) {
        case LIGHT:
            ptlt = (Light *)((uint64_t)(PARAMS.obj_chunk) + light->offset);
            return ptlt->color;
            
        case SPHERE:
            sphere = (Sphere *)((uint64_t)(PARAMS.obj_chunk) + light->offset);
            mat = &(PARAMS.mat_list[sphere->material]);
            return make_float3(mat->color.x * mat->emissive,
                                mat->color.y * mat->emissive,
                                mat->color.z * mat->emissive);
    }

    return make_float3(0.0f, 0.0f, 0.0f);
}

__device__ float3 device::GetRandomLightPosition(hiprandState *rand_state, LightObject *light) {
    Light *ptlt = NULL;
    Sphere *sphere = NULL;

    switch (light->type) {
        case LIGHT:
            ptlt = (Light *)((uint64_t)(PARAMS.obj_chunk) + light->offset);
            return ptlt->position;
            
        case SPHERE:
            sphere = (Sphere *)((uint64_t)(PARAMS.obj_chunk) + light->offset);
            float3 dir = normalize(make_float3(hiprand_uniform(rand_state) - 0.5f,
                                                hiprand_uniform(rand_state) - 0.5f,
                                                hiprand_uniform(rand_state) - 0.5f));
            float r = hiprand_uniform(rand_state) * sphere->radius;
            return sphere->position + make_float3(dir.x * r,
                                                   dir.y * r,
                                                   dir.z * r);
    }
    
    return make_float3(0.0f, 0.0f, 0.0f);
}

__device__ void device::DirectShading(Ray *ray, Intersection *obj) {
    float3 hit_pt = evaluate(ray, obj->t);
    Material *mat = GetMaterial(obj);
    float3 N = Normal(obj, hit_pt);
    float contrib = ray->contrib * 1.0f / PARAMS.num_lights * 1.0f / PARAMS.render.direct_samples;
    float3 clr = {0.0f, 0.0f, 0.0f};
    
    // bring the rand state into local memory for faster access
    hiprandState rand_state = GetRandState();

    // emissive component
    clr.x += ray->contrib * mat->emissive * mat->color.x;
    clr.y += ray->contrib * mat->emissive * mat->color.y;
    clr.z += ray->contrib * mat->emissive * mat->color.z;

    // sample each light direct_samples times
    for (uint64_t i = 0; i < PARAMS.num_lights; i++) {
        LightObject light = PARAMS.light_list[i];
        float3 light_clr = GetLightColor(&light);    
        
        // record each sample
        for (uint32_t j = 0; j < PARAMS.render.direct_samples; j++) {
            float3 light_pos = GetRandomLightPosition(&rand_state, &light);
            float3 L = normalize(light_pos - hit_pt);
            
            // shadow test
            Ray shadow_probe;
            shadow_probe.origin = hit_pt;
            shadow_probe.direction = L;
            shadow_probe.origin = evaluate(&shadow_probe, EPSILON); // prevent self-intersection
            Intersection occluder = NearestObj(&shadow_probe);
            if (occluder.ptr != NULL) {
                // is the occluder between the light and the hit point, and NOT
                // the light itself?
                uint64_t light_ptr = (uint64_t)(PARAMS.obj_chunk) + light.offset;
                if (occluder.t < distance(hit_pt, light_pos) &&
                    (uint64_t)(occluder.ptr) != light_ptr) {
                    // yes it is, move along folks, nothing to see here
                    continue;
                }
            }
            
            // diffuse component
            float NdotL = dot(N, L);
            NdotL = (NdotL > 0.0f) ? NdotL : 0.0f; // clamp to positive contributions only
            clr.x += contrib * mat->diffuse * mat->color.x * light_clr.x * NdotL;
            clr.y += contrib * mat->diffuse * mat->color.y * light_clr.y * NdotL;
            clr.z += contrib * mat->diffuse * mat->color.z * light_clr.z * NdotL;
            
            // specular component (half angle approximation)
            float3 H = normalize(L + (ray->direction * make_float3(-1.0f, -1.0f, -1.0f)));
            float NdotH = dot(N, H);
            NdotH = (NdotH > 0.0f) ? NdotH : 0.0f; // clamp to positive contributions only
            clr.x += contrib * mat->specular * light_clr.x * pow(NdotH, 1.0f / mat->shininess);
            clr.y += contrib * mat->specular * light_clr.y * pow(NdotH, 1.0f / mat->shininess);
            clr.z += contrib * mat->specular * light_clr.z * pow(NdotH, 1.0f / mat->shininess);
        }
    }
    
    // blend with the layer buffer
    BlendWithLayerBuffer(ray->pixel, ray->layer, clr);
}

// ===== kernel functions =====

__global__ void device::InitRandomness(uint64_t seed, hiprandState *rand_states) {
    uint32_t id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // give all threads same seed, different sequence number, no offset
    hiprand_init(seed, id, 0, &(rand_states[id]));
}

__global__ void device::RayTrace(RayPacket packet) {
    // compute which ray this thread should be tracing
    uint32_t ray_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (ray_index >= packet.num_rays) return;
    Ray *ray = &(packet.rays[ray_index]);

    // find the nearest object
    Intersection obj = NearestObj(ray);

    // if the ray hit something...
    if (obj.ptr != NULL) {
        // compute direct lighting
        DirectShading(ray, &obj);

        // TODO: generate importance rays

        // TODO: generate ambient rays
    }
}
